#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void seam_gpu_kernel(const PtrStepSz<uchar3> src1, const PtrStepSz<uchar3> src2, PtrStep<uchar3> dst, const int* seamline)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//int offset = x + y * blockDim.x * gridDim.x;

	if (x < src1.cols && y < src1.rows)
	{
		//uchar3 v = src1(y, x);
		uchar3 v;
		if (x < seamline[y])
			v = src1(y, x);
		else
			v = src2(y, x);
		dst(y, x) = make_uchar3(v.x, v.y, v.z);
	}
}


void seam_gpu_caller(const PtrStepSz<uchar3>& src1, const PtrStepSz<uchar3>& src2, PtrStep<uchar3> dst, const int* seamline, hipStream_t stream)
{
	dim3 block(32, 8);
	dim3 grid((src1.cols + block.x - 1) / block.x, (src1.rows + block.y - 1) / block.y);

	seam_gpu_kernel << <grid, block, 0, stream >> >(src1, src2, dst, seamline);
	if (stream == 0)
		hipDeviceSynchronize();
}


void seam_gpu(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const vector<int>& gseamLine, Stream& stream)
{
	dst.create(src1.size(), src1.type());
	hipStream_t s = StreamAccessor::getStream(stream);

	int sz = gseamLine.size();
	int* seam = new int[sz];

//#pragma omp parallel for
	for (int i = 0; i < sz; i++)
		seam[i] = gseamLine[i];
	int* seamline;
	hipMalloc((void**)&seamline, sz*sizeof(int));
	hipMemcpy(seamline, seam, sz*sizeof(int), hipMemcpyHostToDevice);

	seam_gpu_caller(src1, src2, dst, seamline, s);

	hipFree(seamline);
	free(seam);
}